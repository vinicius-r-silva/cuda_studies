
#include <hip/hip_runtime.h>



#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <ctype.h>
#include <math.h>

#define CEIL(a,b)  ((a+b-1)/b)
#define IPH        ip.Hpixels
#define IPV        ip.Vpixels

#define EDGE 255
#define NOEDGE 0

#define MB(bytes)     (bytes/1024/1024)
#define BW(bytes,timems) ((float)bytes/(timems * 1.024*1024.0*1024.0))

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct ImgProp {
	uint32_t Hpixels;
	uint32_t Vpixels;

	uint8_t HeaderInfo[14];
	uint8_t* HeaderMeta;
	uint16_t HeaderMetaSize;

	uint32_t Hbytes;
	uint32_t IMAGESIZE;
	uint32_t IMAGEPIX;
};

int ThreshLo=50, ThreshHi=100;

struct ImgProp ip;
uint8_t* TheImg, * CpyImg;
uint8_t* GPUImg, * GPUResultImg;
double*  GPUptr, *GPUBWImg, *GPUGaussImg, *GPUGradient, *GPUTheta;

uint8_t* ReadBMPlin(char* fn) {
	static uint8_t* Img;
    FILE* f = fopen(fn, "rb");
	if (f == NULL) { printf("\n\n%s NOT FOUND\n\n", fn); exit(EXIT_FAILURE); }
	uint8_t HeaderInfo[14];
	fread(HeaderInfo, sizeof(uint8_t), 14, f); // read the 54-byte header
	ip.HeaderMetaSize = *(int*)&HeaderInfo[10];
	ip.HeaderMeta = (uint8_t*)malloc(ip.HeaderMetaSize * sizeof(uint8_t));
	fread(ip.HeaderMeta, sizeof(uint8_t), ip.HeaderMetaSize, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&(ip.HeaderMeta[4]); ip.Hpixels = width;
	int height = *(int*)&(ip.HeaderMeta[8]); ip.Vpixels = height;
	//int RowBytes = (width * 3 + 3) & (~3); ip.Hbytes = RowBytes;
	int RowBytes = ip.Hpixels * 3; ip.Hbytes = RowBytes;

	ip.IMAGESIZE = ip.Hbytes * ip.Vpixels;
	ip.IMAGEPIX = ip.Hpixels * ip.Vpixels;
	memcpy(ip.HeaderInfo, HeaderInfo, 14); //save header for re-use
	printf("\n Input File name: %17s\n\nHeaderMetaSize: %u, Hb: %u, Hp: %u, Vp: %u, File Size=%u\n\n", fn,
		ip.HeaderMetaSize, ip.Hbytes, ip.Hpixels, ip.Vpixels, ip.IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img = (uint8_t*)malloc(ip.IMAGESIZE);
	if (Img == NULL) return Img; // Cannot allocate memory
	// read the image from disk
	fread(Img, sizeof(uint8_t), ip.IMAGESIZE, f); fclose(f); return Img;
}

// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uint8_t* Img, char* fn) {
	FILE* f = fopen(fn, "wb");
	if (f == NULL) { printf("\n\nFILE CREATION ERROR: %s\n\n", fn); exit(1); }
	fwrite(ip.HeaderInfo, sizeof(uint8_t), 14, f); //write header
	fwrite(ip.HeaderMeta, sizeof(uint8_t), ip.HeaderMetaSize, f); //write header
	fwrite(Img, sizeof(uint8_t), ip.IMAGESIZE, f); //write data
	printf("\nOutput File name: %17s (%u x %u) File Size=%u\n\n", fn, ip.Hpixels,
		ip.Vpixels, ip.IMAGESIZE);
	fclose(f);
}


__global__
void BWKernel(double *ImgBW, uint8_t *ImgGPU, uint32_t Hpixels){
    uint32_t ThrPerBlk = blockDim.x;
    uint32_t MYbid = blockIdx.x;
    uint32_t MYtid = threadIdx.x;
    uint32_t MYgtid = ThrPerBlk * MYbid + MYtid;
    double R, G, B;

    uint32_t BlkPerRow = CEIL(Hpixels, ThrPerBlk);
    uint32_t RowBytes = Hpixels * 3;
    uint32_t MYrow = MYbid / BlkPerRow;
    uint32_t MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
    if (MYcol >= Hpixels)return; // col out of range
    
    uint32_t MYsrcIndex = MYrow * RowBytes + 3 * MYcol;
    uint32_t MYpixIndex = MYrow * Hpixels + MYcol;

    B = (double)ImgGPU[MYsrcIndex];
    G = (double)ImgGPU[MYsrcIndex + 1];
    R = (double)ImgGPU[MYsrcIndex + 2];
    ImgBW[MYpixIndex] = (R+G+B)/3.0;
}

__device__
double Gauss[5][5]={{ 2,   4,    5,  4,   2 },
                    { 4,  9,   12,  9,   4 },
                    { 5,  12,  15,  12,  5 },
                    { 4,  9,   12,  9,   4 },
                    { 2,  4,   5,   4,   2 } };
// Kernel that calculates a Gauss image from the B&W image
// resulting image has a double type for each pixel position

__global__
void GaussKernel(double *ImgGauss, double*ImgBW, uint32_t Hpixels, uint32_t Vpixels){
    uint32_t ThrPerBlk = blockDim.x;uint32_t MYbid = blockIdx.x;
    uint32_t MYtid = threadIdx.x;
    uint32_t MYgtid = ThrPerBlk * MYbid + MYtid;
    int row, col, indx, i, j;
    double G=0.00;
    
    //uint32_t NumBlocks = gridDim.x;
    uint32_t BlkPerRow = CEIL(Hpixels, ThrPerBlk);
    int MYrow = MYbid / BlkPerRow;
    int MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
    if (MYcol >= Hpixels) return; // col out of range
    
    uint32_t MYpixIndex = MYrow * Hpixels + MYcol;
    
    if ((MYrow<2) || (MYrow>Vpixels - 3) || (MYcol<2) || (MYcol>Hpixels - 3)){
        ImgGauss[MYpixIndex] = 0.0;
        return;
    }else{
        G = 0.0;
        for(i = -2; i <= 2; i++){
            for(j = -2; j <= 2; j++){
                row = MYrow + i;
                col = MYcol + j;
                indx = row*Hpixels + col;
                G += (ImgBW[indx] * Gauss[i + 2][j + 2]);
            }
        }
        ImgGauss[MYpixIndex] = G / 159.00;
    }
}



__device__
double Gx[3][3] = { { -1, 0,  1 },
                    { -2, 0,  2 },
                    { -1, 0,  1 } };

__device__
double Gy[3][3] = { { -1, -2, -1 },
                    {  0,  0,  0 },
                    {  1,  2,  1 } };
// Kernel that calculates Gradient, Theta from the Gauss image
// resulting image has a double type for each pixel position

__global__
void SobelKernel(double *ImgGrad, double*ImgTheta,double*ImgGauss, uint32_t Hpixels,uint32_t Vpixels){
    uint32_t ThrPerBlk = blockDim.x;
    uint32_t MYbid = blockIdx.x;
    uint32_t MYtid = threadIdx.x;
    uint32_t MYgtid = ThrPerBlk * MYbid + MYtid;
    int row, col, indx, i, j;
    double GX,GY;

    //uint32_t NumBlocks = gridDim.x;
    uint32_t BlkPerRow = CEIL(Hpixels, ThrPerBlk);
    int MYrow = MYbid / BlkPerRow;
    int MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
    if (MYcol >= Hpixels)return; // col out of range
    
    uint32_t MYpixIndex = MYrow * Hpixels + MYcol;
    if ((MYrow<1) || (MYrow>Vpixels - 2) || (MYcol<1) || (MYcol>Hpixels - 2)){
        ImgGrad[MYpixIndex] = 0.0;
        ImgTheta[MYpixIndex] = 0.0;
        return;
    }
    else{
        GX = 0.0;
        GY = 0.0;
        for(i = -1; i <= 1; i++){
            for(j = -1; j <= 1; j++){
                row = MYrow + i;
                col = MYcol + j;
                indx = row*Hpixels + col;
                GX += (ImgGauss[indx] * Gx[i + 1][j + 1]);
                GY += (ImgGauss[indx] * Gy[i + 1][j + 1]);
            }
        }
        ImgGrad[MYpixIndex] = sqrt(GX*GX + GY*GY);
        ImgTheta[MYpixIndex] = atan(GX / GY)*180.0 / M_PI;
    }
}

// Kernel that calculates the threshold image from Gradient, Theta
// resulting image has an RGB for each pixel, same RGB for each pixel
__global__
void ThresholdKernel(uint8_t *ImgResult, double* ImgGrad, double* ImgTheta, uint8_t Hpixels, uint32_t Vpixels, uint32_t ThreshLo, uint32_t ThreshHi){
    uint32_t ThrPerBlk= blockDim.x;
    uint32_t MYbid= blockIdx.x;
    uint32_t MYtid= threadIdx.x;
    uint32_t MYgtid= ThrPerBlk*MYbid+MYtid;
    
    double L,H,G,T;
    uint8_t PIXVAL;
    uint32_t BlkPerRow= CEIL(Hpixels,ThrPerBlk);
    
    int MYrow = MYbid / BlkPerRow;
    uint32_t RowBytes= Hpixels*3;
    
    int MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
    if (MYcol >= Hpixels)
        return; // col out of range
    
    uint32_t MYresultIndex= MYrow*RowBytes+3*MYcol;
    uint32_t MYpixIndex= MYrow*Hpixels+MYcol;
    if ((MYrow<1) || (MYrow>Vpixels-2) || (MYcol<1) || (MYcol>Hpixels-2)){
        ImgResult[MYresultIndex]= NOEDGE;
        ImgResult[MYresultIndex+1]= NOEDGE;
        ImgResult[MYresultIndex+2]= NOEDGE;
        return;
    }else{
        L = (double)ThreshLo;
        H = (double)ThreshHi;
        G = ImgGrad[MYpixIndex];
        PIXVAL= NOEDGE;
        
        if (G <= L){
            PIXVAL= NOEDGE; // no edge
        
        }else if(G >= H){
            PIXVAL = EDGE; // edge
        
        }else{
            T = ImgTheta[MYpixIndex];
            if ((T<-67.5) || (T>67.5)){ 
                // Look at left and right: [row][col-1] and [row][col+1]
                PIXVAL= ((ImgGrad[MYpixIndex-1]>H) || (ImgGrad[MYpixIndex+1]>H)) ? EDGE :NOEDGE;
            
            }else if((T >= -22.5) && (T <= 22.5)){
                // Look at top and bottom: [row-1][col] and [row+1][col]
                PIXVAL= ((ImgGrad[MYpixIndex-Hpixels]>H) ||(ImgGrad[MYpixIndex+Hpixels]>H)) ? EDGE : NOEDGE;
            
            }else if((T>22.5) && (T <= 67.5)){
                // Look at upper right, lower left: [row-1][col+1] and [row+1][col-1]
                PIXVAL= ((ImgGrad[MYpixIndex-Hpixels+1]>H) ||(ImgGrad[MYpixIndex+Hpixels-1]>H)) ? EDGE : NOEDGE;
            
            }else if((T >= -67.5) && (T<-22.5)){
                // Look at upper left, lower right: [row-1][col-1] and [row+1][col+1]
                PIXVAL=((ImgGrad[MYpixIndex-Hpixels-1]>H) ||(ImgGrad[MYpixIndex+Hpixels+1]>H)) ? EDGE : NOEDGE;
            }
        }
        ImgResult[MYresultIndex]=PIXVAL;
        ImgResult[MYresultIndex+1]=PIXVAL;
        ImgResult[MYresultIndex+2]=PIXVAL;
    }
}

int main(){ 
    hipEvent_t time1, time2, time2BW, time2Gauss, time2Sobel, time3, time4;
    hipDeviceProp_t GPUprop;
    
    int NumGPUs = 0; hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		std::cout << "\nNo CUDA Device is available\n";
		exit(EXIT_FAILURE);
    }
    
	uint32_t SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;
	char SupportedBlocks[100];

    uint32_t BlkPerRow, ThrPerBlk = 256, NumBlocks;
    uint64_t GPUDataTfrBW, GPUDataTfrGauss, GPUDataTfrSobel, GPUDataTfrThresh;
    float totalKernelTime, totalTime, tfrCPUtoGPU, tfrGPUtoCPU;
    float kernelExecTimeBW, kernelExecTimeGauss, kernelExecTimeSobel, kernelExecTimeThreshold;
    float GPUDataTfrKernel, GPUDataTfrTotal;

	char InputFileName[] = "../img/img.bmp";
    char OutputFileName[] = "../img/edge.bmp";

    TheImg = ReadBMPlin(InputFileName);
	CpyImg = (uint8_t*)malloc(ip.IMAGESIZE);

    hipGetDeviceProperties(&GPUprop, 0);
	SupportedKBlocks = ((uint32_t)GPUprop.maxGridSize[0] * (uint32_t)GPUprop.maxGridSize[1] *
		(uint32_t)GPUprop.maxGridSize[2]) / 1024;

	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%u %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks,
		(SupportedMBlocks >= 5) ? 'M' : 'K');
    MaxThrPerBlk = (uint32_t)GPUprop.maxThreadsPerBlock;
    
	hipEventCreate(&time1); hipEventCreate(&time2);
	hipEventCreate(&time3); hipEventCreate(&time4);
    hipEventCreate(&time2BW); hipEventCreate(&time2Sobel);
    hipEventCreate(&time2Gauss);

    hipEventRecord(time1, 0);
    uint64_t GPUtotalBufferSize = 4 *sizeof(double)*ip.IMAGEPIX + 2 *sizeof(uint8_t)*ip.IMAGESIZE;
    gpuErrchk(hipMalloc((void**)&GPUptr, GPUtotalBufferSize));
    
    GPUImg       = (uint8_t *)GPUptr;
    GPUResultImg = GPUImg + ip.IMAGESIZE;
    GPUBWImg     = (double *)(GPUResultImg + ip.IMAGESIZE);
    GPUGaussImg  = GPUBWImg + ip.IMAGEPIX;
    GPUGradient  = GPUGaussImg + ip.IMAGEPIX;
    GPUTheta     = GPUGradient + ip.IMAGEPIX;

    gpuErrchk(hipMemcpy(GPUImg, TheImg, ip.IMAGESIZE, hipMemcpyHostToDevice));
    hipEventRecord(time2, 0);

    BlkPerRow=CEIL(IPH, ThrPerBlk);
    NumBlocks=IPV*BlkPerRow;

    BWKernel <<< NumBlocks, ThrPerBlk >>> (GPUBWImg, GPUImg, IPH);
    GPUDataTfrBW =sizeof(double)*ip.IMAGEPIX +sizeof(uint8_t)*ip.IMAGESIZE;
    hipEventRecord(time2BW, 0);

    GaussKernel <<< NumBlocks, ThrPerBlk >>> (GPUGaussImg, GPUBWImg, IPH, IPV);
    GPUDataTfrGauss = 2*sizeof(double)*ip.IMAGEPIX;
    hipEventRecord(time2Gauss, 0);// after Gauss image calculation

    SobelKernel <<<  NumBlocks, ThrPerBlk >>> (GPUGradient, GPUTheta, GPUGaussImg, IPH, IPV);
    GPUDataTfrSobel = 3 *sizeof(double)*ip.IMAGEPIX;
    hipEventRecord(time2Sobel, 0);// after Gradient, Theta computation

    ThresholdKernel <<< NumBlocks, ThrPerBlk >>> (GPUResultImg, GPUGradient,GPUTheta, IPH, IPV, ThreshLo, ThreshHi);
    GPUDataTfrThresh=sizeof(double)*ip.IMAGEPIX +sizeof(uint8_t)*ip.IMAGESIZE;
    hipEventRecord(time3, 0);  // after threshold
    
    gpuErrchk(hipMemcpy(CpyImg, GPUResultImg, ip.IMAGESIZE, hipMemcpyDeviceToHost));
    hipEventRecord(time4, 0);  // after GPU-> CPU tfr

    
    gpuErrchk(hipDeviceSynchronize());
    WriteBMPlin(CpyImg, OutputFileName);
    

	hipEventSynchronize(time1); hipEventSynchronize(time2);
	hipEventSynchronize(time3); hipEventSynchronize(time4);
	hipEventSynchronize(time2BW); hipEventSynchronize(time2Gauss);
    hipEventSynchronize(time2Sobel);
    
    hipEventElapsedTime(&totalKernelTime, time2, time3);
    hipEventElapsedTime(&totalTime, time1, time4);
    
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
    hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);
    
    hipEventElapsedTime(&kernelExecTimeBW, time2, time2BW);
	hipEventElapsedTime(&kernelExecTimeGauss, time2BW, time2Gauss);
	hipEventElapsedTime(&kernelExecTimeSobel, time2Gauss, time2Sobel);
	hipEventElapsedTime(&kernelExecTimeThreshold, time2Sobel, time3);
    
    GPUDataTfrKernel = GPUDataTfrBW + GPUDataTfrGauss + GPUDataTfrSobel + GPUDataTfrThresh;
    GPUDataTfrTotal  = GPUDataTfrKernel + 2 * ip.IMAGESIZE;


    printf("\n\n---------------------\n");
    printf("%s   ComputeCapab=%d.%d [max %s blocks; %d thr/blk] \n",GPUprop.name, GPUprop.major, GPUprop.minor, SupportedBlocks, MaxThrPerBlk);
    printf("\n\n---------------------\n");
    printf("%s %s %u %d %d [%u BLOCKS, %u BLOCKS/ROW]\n", InputFileName, OutputFileName, ThrPerBlk, ThreshLo, ThreshHi, NumBlocks,BlkPerRow);
    printf("\n\n---------------------\n");
    printf("CPU->GPU Transfer = %f ms ... %ui MB ... %f GB/s\n", tfrCPUtoGPU, MB(ip.IMAGESIZE),BW(ip.IMAGESIZE,tfrCPUtoGPU));
    printf("GPU->CPU Transfer = %f ms ... %ui MB ... %f GB/s\n", tfrGPUtoCPU, MB(ip.IMAGESIZE),BW(ip.IMAGESIZE, tfrGPUtoCPU));
    printf("\n\n---------------------\n");
    printf("     BW Kernel Execution Time = %f ms ... %lu MB ... %f GB/s\n", kernelExecTimeBW,MB(GPUDataTfrBW), BW(GPUDataTfrBW, kernelExecTimeBW));
    printf("   Gauss Kernel Execution Time = %f ms ... %lu MB ... %f GB/s\n", kernelExecTimeGauss,MB(GPUDataTfrGauss), BW(GPUDataTfrGauss, kernelExecTimeGauss));
    printf("   Sobel Kernel Execution Time = %f ms ... %lu MB ... %f GB/s\n", kernelExecTimeSobel,MB(GPUDataTfrSobel), BW(GPUDataTfrSobel, kernelExecTimeSobel));
    printf("Threshold Kernel Execution Time = %f ms ... %lu MB ... %f GB/s\n", kernelExecTimeThreshold,MB(GPUDataTfrThresh), BW(GPUDataTfrThresh, kernelExecTimeThreshold));
    printf("\n\n---------------------\n");
    printf("       Total Kernel-only time = %f ms ... %lf MB ... %f GB/s\n", totalKernelTime,MB(GPUDataTfrKernel), BW(GPUDataTfrKernel, totalKernelTime));
    printf("  Total time with I/O included = %f ms ... %lf MB ... %f GB/s\n", totalTime, MB(GPUDataTfrTotal),BW(GPUDataTfrTotal, totalTime));
    printf("\n\n---------------------\n");


    return 0;

}