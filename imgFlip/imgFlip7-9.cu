#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <ctype.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>

// #include "cudaDefines.h"

#define REPETITON 1000

struct ImgProp {
	uint32_t Hpixels;
	uint32_t Vpixels;

	uint8_t HeaderInfo[14];
	uint8_t* HeaderMeta;
	uint16_t HeaderMetaSize;

	uint32_t Hbytes;
	uint32_t Hints;
	uint32_t IMAGESIZE;
	uint32_t ARRAYSIZE;
	uint32_t IMAGEPIX;
};

struct ImgProp ip;
uint32_t* TheImg, * CpyImg;
uint32_t* GPUImg, * GPUCopyImg;

cv::Mat Image;

//-----------------------------------------------------------------------------------------------------------------//
//-----------------------------------------------IMG READ WRITE----------------------------------------------------//
uint8_t* ReadBMPlin(char* fn) {
    Image = cv::imread(fn);

    if(!Image.data){                              // Check for invalid input
        std::cout <<  "Could not open or find the image" << std::endl ;
        exit(EXIT_FAILURE);
    }

    // cv::resize(Image, Image, cv::Size(0,0), 0.5, 0.5);
    ip.Hpixels = Image.cols;
    ip.Vpixels = Image.rows;
	ip.Hbytes = Image.cols * 3;
	ip.Hints = ip.Hbytes / 4;
    
	ip.IMAGESIZE = ip.Hbytes * ip.Vpixels;
	ip.IMAGEPIX = ip.Hpixels * ip.Vpixels;
	ip.ARRAYSIZE = ip.IMAGESIZE/4;
    
    return Image.data;
}

// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uint8_t* Img, char* fn) {
    memcpy(Image.data, Img, ip.IMAGESIZE);
    imwrite(fn, Image);
}




//-----------------------------------------------------------------------------------------------------------------//
//-----------------------------------------------------VFLIP-------------------------------------------------------//
__global__
void Vflip7(uint32_t* ImgDst, uint32_t* ImgSrc, uint32_t Hints, uint32_t Vpixels) {
	__shared__ uint32_t PixBuffer[1024];

	uint32_t ThrPerBlk = blockDim.x;
	uint32_t MYbid = blockIdx.x;
	uint32_t MYtid = threadIdx.x;
	
	uint32_t MYrow = blockIdx.y;
	uint32_t MYcol = MYbid * ThrPerBlk + MYtid;
	if (MYcol >= Hints) return; // col out of range

	uint32_t MYmirrorrow = Vpixels - 1 - MYrow;
	uint32_t MYsrcOffset = MYrow * Hints;
	uint32_t MYdstOffset = MYmirrorrow * Hints;
	uint32_t MYsrcIndex = MYsrcOffset + MYcol;
	uint32_t MYdstIndex = MYdstOffset + MYcol;

	PixBuffer[MYtid] = ImgSrc[MYsrcIndex];
	__syncthreads();
	ImgDst[MYdstIndex] = PixBuffer[MYtid];	
}


__global__
void Vflip8(uint32_t* ImgDst, uint32_t* ImgSrc, uint32_t Hints, uint32_t Vpixels) {
	__shared__ uint32_t PixBuffer[1024];

	uint32_t ThrPerBlk = blockDim.x;
	uint32_t MYbid = blockIdx.x;
	uint32_t MYtid = threadIdx.x;
	
	uint32_t MYrow = blockIdx.y;
	uint32_t MYcol = (MYbid * ThrPerBlk + MYtid)*2;
	if (MYcol >= Hints) return; // col out of range
	MYcol++;

	uint32_t MYmirrorrow = Vpixels - 1 - MYrow;
	uint32_t MYsrcOffset = MYrow * Hints;
	uint32_t MYdstOffset = MYmirrorrow * Hints;
	uint32_t MYsrcIndex = MYsrcOffset + MYcol;
	uint32_t MYdstIndex = MYdstOffset + MYcol;

	PixBuffer[MYtid] = ImgSrc[MYsrcIndex];
	if(MYcol < Hints) PixBuffer[MYtid + 1] = ImgSrc[MYsrcIndex + 1];
	__syncthreads();
	ImgDst[MYdstIndex] = PixBuffer[MYtid];	
	if(MYcol < Hints) ImgDst[MYdstIndex + 1] = PixBuffer[MYtid + 1];
}


__global__
void Vflip9(uint32_t* ImgDst, uint32_t* ImgSrc, uint32_t Hints, uint32_t Vpixels) {
	// __shared__ uint32_t PixBuffer[1024];

	uint32_t ThrPerBlk = blockDim.x;
	uint32_t MYbid = blockIdx.x;
	uint32_t MYtid = threadIdx.x;
	
	uint32_t MYrow = blockIdx.y;
	uint32_t MYcol = (MYbid * ThrPerBlk + MYtid)*2;
	if (MYcol >= Hints) return; // col out of range
	MYcol++;

	uint32_t MYmirrorrow = Vpixels - 1 - MYrow;
	uint32_t MYsrcOffset = MYrow * Hints;
	uint32_t MYdstOffset = MYmirrorrow * Hints;
	uint32_t MYsrcIndex = MYsrcOffset + MYcol;
	uint32_t MYdstIndex = MYdstOffset + MYcol;

	ImgDst[MYdstIndex] = ImgSrc[MYsrcIndex];
	if(MYcol < Hints) ImgDst[MYdstIndex + 1] = ImgSrc[MYsrcIndex + 1];
}



//-----------------------------------------------------------------------------------------------------------------//
//-----------------------------------------------------MAIN--------------------------------------------------------//
int main(int argc, char *argv[]){
	hipError_t cudaStatus, cudaStatus2;
	hipEvent_t time1, time2, time3, time4;
	hipDeviceProp_t GPUprop;

	int KernelNumber = 1;
	if(argc > 1)
		KernelNumber = atoi(argv[1]);

	uint32_t SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;
	char SupportedBlocks[100];

	uint32_t BlkPerRow, ThrPerBlk = 128, NumBlocks, GPUDataTransfer;
	float totalTime, tfrCPUtoGPU, kernelExecutionTime, tfrGPUtoCPU;

	char InputFileName[] = "../img/img.bmp";
	char OutputFileName[] = "../img/flip.bmp";
	char KernelName[100]; memset(KernelName, '\0', 100);

	TheImg = (uint32_t*)ReadBMPlin(InputFileName);
	CpyImg = (uint32_t*)malloc(ip.IMAGESIZE);
	//WriteBMPlin(TheImg, OutputFileName);
	//return 0;

	int NumGPUs = 0; hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0) {
		std::cout << "\nNo CUDA Device is available\n";
		exit(EXIT_FAILURE);
	}

	hipGetDeviceProperties(&GPUprop, 0);
	SupportedKBlocks = ((uint32_t)GPUprop.maxGridSize[0] * (uint32_t)GPUprop.maxGridSize[1] *
		(uint32_t)GPUprop.maxGridSize[2]) / 1024;

	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%u %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks,
		(SupportedMBlocks >= 5) ? 'M' : 'K');
	MaxThrPerBlk = (uint32_t)GPUprop.maxThreadsPerBlock;

	hipEventCreate(&time1); hipEventCreate(&time2);
	hipEventCreate(&time3); hipEventCreate(&time4);

	BlkPerRow = (ip.Hints + ThrPerBlk - 1) / ThrPerBlk;
	NumBlocks = ip.Vpixels * BlkPerRow;
	GPUDataTransfer = 2 * ip.IMAGESIZE;

	hipEventRecord(time1, 0);
	cudaStatus = hipMalloc((void**)&GPUImg, ip.IMAGESIZE);
	cudaStatus2 = hipMalloc((void**)&GPUCopyImg, ip.IMAGESIZE);
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess)) {
		std::cout << "hipMalloc failed! Can't allocate GPU memory";
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipMemcpy(GPUImg, TheImg, ip.IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		std::cout << "cudaMemCpy CPU to GPU failed!";
		exit(EXIT_FAILURE);
	}


	// uint32_t RowBytes = (ip.Hpixels * 3 + 3) & (~3);
	// uint32_t RowBytes = ip.Hbytes;
	int i = 0;
	hipEventRecord(time2, 0);

	if(KernelNumber == 7){
		dim3 dimGrid2D(BlkPerRow, ip.Vpixels);
		for(i = 0; i < REPETITON; i++){
			Vflip7 <<<dimGrid2D, ThrPerBlk>>> (GPUCopyImg, GPUImg, ip.Hints, ip.Vpixels);
		}
		strcpy(KernelName,"VFlip7: Each thread copies 1 pixel (using a 2D grid)");
	}
	else if(KernelNumber == 8){
		BlkPerRow = (BlkPerRow + 2 -1) / 2;
		dim3 dimGrid2D(BlkPerRow, ip.Vpixels);
		for(i = 0; i < REPETITON; i++){
			Vflip8 <<<dimGrid2D, ThrPerBlk>>> (GPUCopyImg, GPUImg, ip.Hints, ip.Vpixels);
		}
		strcpy(KernelName,"VFlip8: Each thread copies 1 pixel (using a 2D grid)");
	}
	else{
		printf("Unkown Kernel Number: %d\n", KernelNumber);

		hipFree(GPUImg); hipFree(GPUCopyImg);
		hipEventDestroy(time1); hipEventDestroy(time2);
		hipEventDestroy(time3); hipEventDestroy(time4);
	
		cudaStatus = hipDeviceReset();
		free(CpyImg);
		return 1;		
	}


	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "hipDeviceSynchronize error code " << cudaStatus << " ...\n";
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	cudaStatus = hipMemcpy(CpyImg, GPUCopyImg, ip.IMAGESIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		std::cout << "cudaMemCpy GPU to CPU failed!" << cudaStatus;
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1); hipEventSynchronize(time2);
	hipEventSynchronize(time3); hipEventSynchronize(time4);
	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	kernelExecutionTime /= REPETITON;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Program failed afeter cudaDeviceSyncronize()";
		free(TheImg); free(CpyImg);
		exit(EXIT_FAILURE);
	}

	WriteBMPlin((uint8_t*)CpyImg, OutputFileName);

	printf("--...--\n"); 
	
	printf("%s\n", KernelName);

	// printf("%s ComputeCapab=%d.%d [supports max %s blocks]\n",
	// 	GPUprop.name, GPUprop.major, GPUprop.minor, SupportedBlocks); printf("...\n");
	printf("maxTrPerBlk: %d\n", MaxThrPerBlk);
	printf("%s %u x %u\n%s\n\nThrPerBlock: %u, Blocks: %u, BlkPerRow: %u\n", InputFileName, ip.Hpixels, ip.Vpixels,
		OutputFileName, ThrPerBlk, NumBlocks, BlkPerRow);
	printf("-------------------- ... ----------------------------\n");
	printf("CPU->GPU Transfer = %5.2f ms ... %4d MB ... %6.2f GB/s\n",
		tfrCPUtoGPU, ip.IMAGESIZE / 1024 / 1024, (float)ip.IMAGESIZE / (tfrCPUtoGPU *
			1024.0 * 1024.0));
	printf("Kernel Execution = %5.2f ms ... %4d MB ... %6.2f GB/s (%3.2f%%)\n",
		kernelExecutionTime, GPUDataTransfer / 1024 / 1024, (float)GPUDataTransfer /
		(kernelExecutionTime * 1024.0 * 1024.0), float((float)GPUDataTransfer / (kernelExecutionTime * 1024.0 * 1024.0)) / 1.1210);
	printf("GPU->CPU Transfer = %5.2f ms ... %4d MB ... %6.2f GB/s\n",
		tfrGPUtoCPU, ip.IMAGESIZE / 1024 / 1024, (float)ip.IMAGESIZE / (tfrGPUtoCPU *
			1024.0 * 1024.0));
	printf("Total time elapsed = %5.2f ms\n", totalTime);
	printf("-------------------- ... ----------------------------\n");


	hipFree(GPUImg); hipFree(GPUCopyImg);
	hipEventDestroy(time1); hipEventDestroy(time2);
	hipEventDestroy(time3); hipEventDestroy(time4);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		std::cout << "hipDeviceReset failed!";
		free(TheImg); free(CpyImg); exit(EXIT_FAILURE);
	}
	//free(TheImg); 
	free(CpyImg);

	// getchar();
	//getchar();
	return(EXIT_SUCCESS);
}